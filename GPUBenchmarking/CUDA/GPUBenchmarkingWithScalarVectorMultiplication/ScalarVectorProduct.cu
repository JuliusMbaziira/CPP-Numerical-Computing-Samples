/* 
 * 
 * GPU Computing: Benchmarking CUDA with the Scalar-Vector Product 
 * Computes only one element in a single thread and uses N threads with 1
 * Single Thread-Per-Block.
 * 
 */

#include <stdio.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define SCALAR 3

float printRunTime(const char *string, struct timespec *ti1, struct timespec *ti2)
{
   float runtime;
   
   runtime = (ti2->tv_sec - ti1->tv_sec ) + 1e-9*(ti2->tv_nsec - ti1->tv_nsec);
   fprintf(stderr,"Run time %s : %f microsecs.\n", string, runtime * 1e6);
   return runtime;
}

void show(float *a, long n)
{
   int i,j;
   int max = 10;
   
   j = (n > max) ? n-max : 0;
   
   printf("Results (max last %d) : \n",max);
   
   for(i=j; i<n; i++)
     printf("%3d : %6.2f\n",i,a[i]);
}

// This runs on the CPU :
float multiplyOnHost(float *a, long n, float s)
{
   long i;
   struct timespec ti1,ti2;
   
   fprintf(stderr,"Now computing on CPU : \n");
   clock_gettime(CLOCK_REALTIME,&ti1);        // read starttime into t1
   
   for(i=0;i<n;i++)
     a[i] *= s;
   
   clock_gettime(CLOCK_REALTIME,&ti2);        // read endtime into t2
   return printRunTime("CPU",&ti1,&ti2);
}

// This is the kernel that runs on the GPU :
__global__ void multiplyKernel(float *b, float s)
{
   int i = blockIdx.x * blockDim.x + threadIdx.x;
   b[i] *= s;
}

// This part also runs on the CPU (and not on the GPU !) :
float multiplyOnDevice(float *b_h, long n, float s)
{
   float *a_d;   // Pointer to device array
   struct timespec ti1,ti2,ti3,ti4;
   int nBlocks;
   
   // Do the actual calculation on N threads inside the GPU device:
   nBlocks = n;
   if(nBlocks > 65535)
   {
      fprintf(stderr,"Maximum number of blocks exceeded : %d > 65535 !\n", nBlocks);
      exit(1);
   }
   fprintf(stderr,"Now computing on GPU (%d blocks, %d threads per block): \n", nBlocks, 1);
   
   hipMalloc((void **) &a_d, n * sizeof(float)); // Allocate array on device

   clock_gettime(CLOCK_REALTIME,&ti1);            // read starttime into t1
   hipMemcpy(a_d, b_h, n * sizeof(float), hipMemcpyHostToDevice);
   
   clock_gettime(CLOCK_REALTIME,&ti2);            // read starttime into t2
   multiplyKernel <<< nBlocks, 1 >>> (a_d, s);
   hipDeviceSynchronize();                       // We make sure all threads finished before stopping clock
   
   clock_gettime(CLOCK_REALTIME,&ti3);            // read endtime into t3
   
   // Retrieve result from device and store it in host array
   hipMemcpy(b_h, a_d, sizeof(float)*n, hipMemcpyDeviceToHost);

   clock_gettime(CLOCK_REALTIME,&ti4);            // read endtime into t4
   hipFree(a_d);
   
   printRunTime("GPU (no data transfer)",&ti2,&ti3);

   return printRunTime("GPU (including data transfer)",&ti1,&ti4);
}

// main routine that executes on the host
int main(int argc, char **argv)
{
   float *a_h,*b_h;      // Pointers to host arrays
   long n = 4096L;       // Number of elements in arrays (problemsize)
   long i;
   float ts,tp;
   float s = SCALAR;
   
   if(argc >=2 )
     sscanf(argv[1],"%ld",&n);

   fprintf(stderr,"Starting with n=%ld ....\n",n);
   
   // Allocate arrays on host
   a_h = (float *)malloc(n * sizeof(float));
   b_h = (float *)malloc(n * sizeof(float));

   // Initialize host arrays 
   for (i=0; i<n; i++)
     b_h[i] = a_h[i] = (float)i;
   
   // Run sequential algorithm on CPU
   ts = multiplyOnHost(a_h, n, s);
   
   // Run parallel algorithm on GPU
   tp = multiplyOnDevice(b_h, n, s);
   
   // check results
#if (DEBUG > 0)
   show(a_h,n);

   printf("Comparing results ....\n");
   int diffresults=0;
   for (i=0; i<n; i++)
   {
      float f = fabs(a_h[i] - b_h[i]);
      if(f > 1e-6)
      {	
	 printf("CPU: %f , GPU: %f , diff: %f\n",a_h[i],b_h[i],f);
	 diffresults=1;
      }
   }
   if(diffresults == 0)
      printf("Both results are equal up to 1e-6\n");
#endif
   
   printf("Ratio execution time GPU/CPU: %f\n",tp/ts);

   free(a_h);
   free(b_h);
   exit(0);
}

